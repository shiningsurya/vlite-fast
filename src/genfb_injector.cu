#include "hip/hip_runtime.h"
/* GENFB test for injection
 * To generate filterbank to write to PSRDADA buffer.
 - Defaults:
 - written to output dada buffer
 - kurtosis happens
 - pscrunching happens
 * 
 *
 */


// system support
#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
#include <string.h>
#include <time.h>
#include <fcntl.h>

#include "vdifio.h"

// psrdada support
#include "dada_def.h"
#include "dada_hdu.h"
#include "ipcio.h"
#include "ascii_header.h"
#include "multilog.h"

// local support
#include "process_baseband.h"
#include "util.h"
#include "cuda_util.h"
#include "injection.h"

// from Julia's code
extern "C" {
#include "utils.h"
#include "def.h"
#include "executor.h"
#include "multicast.h"
}

// rt profiling
//#define RT_PROFILE 1

// fighting against decimation

static volatile int NBIT = 8;
static FILE* logfile_fp = NULL;
static FILE* fb_fp = NULL;
static multilog_t* mlog = NULL;
static dada_hdu_t* hdu_out = NULL;
static int mc_control_sock = 0;
static float EARLY = 0.0f;

static double UTCEPOCH = 831297600.0;
static double MJD      = 50208.5;

void usage ()
{
  fprintf (stdout,"Usage: genfb [options]\n"
      "-k hexadecimal shared memory key for output (default: 40)\n"
      "-o print logging messages to stdout (as well as logfile)\n"
      "-b reduce output to b bits (2, 4, and 8[def] are supported))\n"
      "-s seed for random number generator [long; default=42]\n"
      "-e seconds of whitenoise data before anything starts [default=3.0f]\n"
      "-E UTC start epoch of the observation[default=123456789, 29/11/1973]\n"
      "-m MJD of the start epoch[default matched to UTC]\n"
      "-f write filterbank [def=no]\n"
      "-g run on specified GPU\n");
  //"-m retain MUOS band\n"
  //	  ,(uint64_t)READER_SERVICE_PORT);
}

void cleanup (void)
{
  fprintf (stderr,"called cleanup! [GENFB]\n");
  fflush (stderr);
  if (fb_fp) fclose (fb_fp);
  fprintf (stderr,"h1\n");
  fflush (stderr);
  if (hdu_out != NULL)
  {
    dada_hdu_disconnect (hdu_out);
    dada_hdu_destroy (hdu_out);
  }
  fprintf (stderr,"h2\n");
  fflush (stderr);
  if (mc_control_sock > 0)
    shutdown (mc_control_sock, 2);
  fprintf (stderr,"h3\n");
  fflush (stderr);
  if (mlog) {
    multilog (mlog, LOG_INFO, "Completed shutdown [GENFB].\n");
    multilog_close (mlog);
  }
  fprintf (stderr,"h4\n");
  fflush (stderr);
  if (logfile_fp) fclose (logfile_fp);
  fprintf (stderr,"h5\n");
  fflush (stderr);
}

void exit_handler (void) {
  fprintf (stderr, "exit handler called\n");
  fflush (stderr);
  cleanup ();
}

void sigint_handler (int dummy) {
  cleanup ();
  exit (EXIT_SUCCESS);
}

/********************
  Header contains the following 
  which should be provided somehow
  - STATIONID int
  - BEAM int
  - RA  double
  - DEC double
  - NAME char[]
  - SCANSTART double
  - NCHANS int
  - BANDWIDTH double
  - CFREQ double
  - NPOL int
  - NBIT int
  - TSAMP double
  - UTC_START char[]
  - UNIXEPOCH double
 *******************/

// injection_t struct has start which is when the signal should begin. 
// start argument is when the data begins
// because we would want to have white noise preceding the trigger.
int write_psrdada_header (dada_hdu_t* hdu, char* fb_file)
{
  // update the time with the actual data start, since we have discarded
  // some data to reach a 1s boundary
  time_t epoch_seconds = UTCEPOCH;
  struct tm utc_time;
  gmtime_r (&epoch_seconds, &utc_time);
  char dada_utc[DADA_TIMESTR_LENGTH];
  strftime (dada_utc, DADA_TIMESTR_LENGTH, DADA_TIMESTR, &utc_time);

  // initialize observation parameters for filterbank
  // NB the data are upper sideband, so negative channel bandwidth
  double chbw = -BW / NCHAN;
  double tsamp = double(NFFT)/VLITE_RATE*NSCRUNCH*1e6; // NB in mus
  int nchan = CHANMAX-CHANMIN+1;
  double bw = nchan*chbw;
  double freq0 = FCH1;
  double freq = freq0 + 0.5*(CHANMIN+CHANMAX-1)*chbw;

  fprintf (stderr, "before lock\n");
  dadacheck (dada_hdu_lock_write (hdu));
  fprintf (stderr, "after lock\n");
  char* ascii_hdr = ipcbuf_get_next_write (hdu->header_block);
  fprintf (stderr, "after next write\n");
  // default options
  dadacheck (ascii_header_set (ascii_hdr, "STATIONID", "%d", STATIONID));
  dadacheck (ascii_header_set (ascii_hdr, "BEAM", "%d", STATIONID));
  dadacheck (ascii_header_set (ascii_hdr, "RA", "%lf", 0.0));
  dadacheck (ascii_header_set (ascii_hdr, "DEC", "%lf", 0.0));
  dadacheck (ascii_header_set (ascii_hdr, "NAME", "%s", "INJECTED"));
  dadacheck (ascii_header_set (ascii_hdr, "NCHAN", "%d", nchan) );
  dadacheck (ascii_header_set (ascii_hdr, "BANDWIDTH", "%lf", bw) );
  dadacheck (ascii_header_set (ascii_hdr, "CFREQ", "%lf", freq) );
  dadacheck (ascii_header_set (ascii_hdr, "NPOL", "%d", 1) );
  dadacheck (ascii_header_set (ascii_hdr, "NBIT", "%d", NBIT) );
  dadacheck (ascii_header_set (ascii_hdr, "TSAMP", "%lf", tsamp) );
  dadacheck (ascii_header_set (ascii_hdr, "UTC_START", "%s", dada_utc) );

  // changing options
  dadacheck (ascii_header_set (ascii_hdr, "SCANSTART", "%lf", UTCEPOCH));
  dadacheck (ascii_header_set (ascii_hdr, "UNIXEPOCH", "%lf", UTCEPOCH) );
  // also record the VDIF MJD info, this is useful for finding
  // transients in the baseband stream.
  dadacheck (ascii_header_set (ascii_hdr, "VDIF_MJD", "%d", MJD) );
  unsigned long imjd = (MJD - (int)MJD)*86400.0f;
  dadacheck (ascii_header_set (ascii_hdr, "VDIF_SEC", "%lu", imjd) );

  if (fb_file)
    dadacheck (ascii_header_set (ascii_hdr, "SIGPROC_FILE", "%s", fb_file) );
  multilog (hdu->log, LOG_INFO, "%s",ascii_hdr);
  ipcbuf_mark_filled (hdu->header_block, 4096);
  return 0;
}

void write_sigproc_header (FILE* output_fp)
{
  double chbw = -64./NCHAN;
  double tsamp = double(NFFT)/VLITE_RATE*NSCRUNCH;
  // write out a sigproc header
  send_string ("HEADER_START",output_fp);
  send_string ("source_name",output_fp);
  send_string ("INJECTED",output_fp);
  send_int ("barycentric",0,output_fp);
  send_int ("telescope_id",STATIONID,output_fp);
  send_double ("src_raj",0.0,output_fp);
  send_double ("src_dej",0.0,output_fp);
  send_int ("data_type",1,output_fp);
  //
  send_double ("fch1",384+(CHANMIN-0.5)*chbw,output_fp);
  send_double ("foff",chbw,output_fp);//negative foff, fch1 is highest freq
  send_int ("nchans",CHANMAX-CHANMIN+1,output_fp);
  send_int ("nbits",NBIT,output_fp);
  send_double ("tstart",MJD,output_fp);
  send_double ("tsamp",tsamp,output_fp);//[sec]
  send_int ("nifs",1,output_fp);
  send_string ("HEADER_END",output_fp);
}

void get_fbfile (char* fbfile, ssize_t fbfile_len)
{
  // Open up filterbank file using timestamp and antenna
  char currt_string[128];
  time_t epoch_seconds = UTCEPOCH;
  struct tm utc_time;
  gmtime_r (&epoch_seconds, &utc_time);
  strftime (currt_string,sizeof(currt_string), "%Y%m%d_%H%M%S", &utc_time);
  *(currt_string+15) = 0;
  if (CHANMIN < 2411)
    snprintf (fbfile,fbfile_len,"%s/%s_muos_ea%02d.fil",DATADIR,currt_string,STATIONID);
  else
    snprintf (fbfile,fbfile_len,"%s/%s_ea%02d.fil",DATADIR,currt_string,STATIONID);
}

void check_buffer (dada_hdu_t* hdu, multilog_t* log)
{
  ipcbuf_t* buf = (ipcbuf_t*) hdu->data_block;
  uint64_t m_nbufs = ipcbuf_get_nbufs (buf);
  uint64_t m_full_bufs = ipcbuf_get_nfull (buf);
  if (m_full_bufs == (m_nbufs - 1))
  {
    fprintf (stderr,"failed buffer check\n");
    fflush (stderr);
    dadacheck (dada_hdu_unlock_write (hdu));
    multilog (mlog, LOG_ERR,
        "Only one free buffer left!  Aborting output.\n");
    exit (EXIT_FAILURE);
  }
}

void check_ipcio_write (dada_hdu_t* hdu, char* buf, size_t to_write, multilog_t* log)
{
  size_t written = ipcio_write (hdu->data_block,buf,to_write);
  if (written != to_write)
  {
    fprintf (stderr, "failed ipcio write\n"); 
    fflush (stderr);
    multilog (mlog, LOG_ERR, "Tried to write %lu bytes to psrdada buffer but only wrote %lu.", to_write, written);
    exit (EXIT_FAILURE);
  }
}

int main (int argc, char *argv[])
{
  // register SIGINT handling
  signal (SIGINT, sigint_handler);

  // register exit function
  atexit (exit_handler);

  int exit_status = EXIT_SUCCESS;
  key_t key_out = 0x40;
  int stdout_output = 0;
  int write_fb = 0;
  int npol = 1;
  int gpu_id = 0;
  size_t maxn = 0;
  long seed = 42;

  int arg = 0;
  while ((arg = getopt(argc, argv, "hk:of:b:s:e:g:E:m:")) != -1) {
    switch (arg) {

      case 'h':
        usage ();
        return 0;

      case 'k':
        if (sscanf (optarg, "%x", &key_out) != 1) {
          fprintf (stderr, "genfb: could not parse key from %s\n", optarg);
          return -1;
        }
        break;

      case 's':
        if (sscanf (optarg, "%li", &seed) != 1) {
          fprintf (stderr, "genfb: could not read seed from %s\n", optarg);
          return -1;
        }
        break;

      case 'o':
        stdout_output = 1;
        break;

      case 'f':
        if (sscanf (optarg, "%d", &write_fb) != 1) {
          fprintf (stderr, "genfb: could not parse write mode %s\n", optarg);
          return -1;
        }
        break;

      case 'b':
        if (sscanf (optarg, "%d", &NBIT) != 1) {
          fprintf (stderr, "genfb: could not parse number of bits %s\n", optarg);
          return -1;
        }
        if (!(NBIT==2 || NBIT==4 || NBIT==8)) {
          fprintf (stderr, "Unsupported NBIT!\n");
          return -1;
        }
        break;

      case 'e':
        if (sscanf (optarg, "%f", &EARLY) != 1) {
          fprintf (stderr, "genfb: could not parse early %s\n", optarg);
          return -1;
        }
        break;

      case 'g':
        if (sscanf (optarg, "%d", &gpu_id) != 1) {
          fprintf (stderr, "writer: could not parse GPU id %s\n", optarg);
          return -1;
        }
        if (!(gpu_id==0 || gpu_id==1)) {
          fprintf (stderr, "Unsupported GPU id!\n");
          return -1;
        }
        break;
      case 'E':
        if (sscanf (optarg, "%lf", &UTCEPOCH) != 1) {
          fprintf (stderr, "genfb: could not parse UTCEPOCH: %s\n", optarg);
          return -1;
        }

      case 'm':
        if (sscanf (optarg, "%lf", &MJD) != 1) {
          fprintf (stderr, "genfb: could not parse MJD: %s\n", optarg);
          return -1;
        }
    }

  }

  cudacheck (hipSetDevice (gpu_id));
  printf ("Setting CUDA device to %d.\n",gpu_id);
  int nsms;
  hipDeviceGetAttribute (&nsms,hipDeviceAttributeMultiprocessorCount,gpu_id);

  struct timespec ts_1ms = get_ms_ts (1);
  struct timespec ts_10s = get_ms_ts (10000);

#if RT_PROFILE
  hipEvent_t rt_start, rt_stop; //,start_total,stop_total;
  hipEventCreate (&rt_start);
  hipEventCreate (&rt_stop);

  float measured_time=0;
  float read_time=0, proc_time=0, write_time=0, rt_elapsed=0;
#endif

#if PROFILE
  // support for measuring run times of parts
  hipEvent_t start,stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);
  float alloc_time=0, hdr_time=0, read_time=0, todev_time=0, 
        convert_time=0, kurtosis_time=0, fft_time=0, histo_time=0,
        normalize_time=0, tscrunch_time=0, pscrunch_time=0, 
        digitize_time=0, write_time=0, flush_time=0, misc_time=0,
        elapsed=0;
#endif
  // measure full run time time
  hipEvent_t obs_start, obs_stop;
  hipEventCreate (&obs_start);
  hipEventCreate (&obs_stop);

  multilog_t* mlog = multilog_open ("genfb",0);
  if (stdout_output)
    multilog_add (mlog, stdout);

  // sanity checks on configuration parameters
  if (NKURTO != 250 && NKURTO != 500) {
    multilog (mlog, LOG_ERR, "Only NKURTO==250 or 500 supported.\n");
    exit (EXIT_FAILURE);
  }

  // connect to output buffer
  if (key_out) {
    hdu_out = dada_hdu_create (mlog);
    dada_hdu_set_key (hdu_out,key_out);
    if (dada_hdu_connect (hdu_out) != 0) {
      multilog (mlog, LOG_ERR, 
          "Unable to connect to outgoing PSRDADA buffer!\n");
      exit (EXIT_FAILURE);
    }
  }

#if PROFILE
  hipEventRecord(start,0);
#endif

  // voltage samples in a chunk, 2*VLITE_RATE/SEG_PER_SEC (both pols)
  size_t samps_per_chunk = 2*VLITE_RATE/SEG_PER_SEC;
  // FFTs per processing chunk (2 per pol)
  int fft_per_chunk = 2*FFTS_PER_SEG;

  // Only one side fft
  hipfftHandle plan;
  cufftcheck (hipfftPlan1d (&plan,NFFT,HIPFFT_R2C,fft_per_chunk));

  // memory for FFTs
  hipfftReal* fft_in; cudacheck (
      hipMalloc ((void**)&fft_in,sizeof(hipfftReal)*samps_per_chunk) );
  hipfftComplex* fft_out; cudacheck (
      hipMalloc ((void**)&fft_out,sizeof(hipfftComplex)*fft_per_chunk*NCHAN) );
  hipfftReal* fft_in_kur; cudacheck ( 
      hipMalloc ((void**)&fft_in_kur,sizeof(hipfftReal)*samps_per_chunk) );

  // device memory for kurtosis statistics, uses NKURTO samples, both pols
  // NKURTO must be commensurate with samps_per_chunk/2, i.e. samples per
  // chunk in a pol
  size_t nkurto_per_chunk = samps_per_chunk / NKURTO;

  // extra factor of 2 to store both power and kurtosis statistics
  // storage for high time resolution and filterbank block ("fb") scales
  hipfftReal *pow_dev(NULL), *kur_dev(NULL), 
            *pow_fb_dev(NULL), *kur_fb_dev(NULL);
  cudacheck (
      hipMalloc ((void**)&pow_dev,2*sizeof(hipfftReal)*nkurto_per_chunk) );
  cudacheck (
      hipMalloc ((void**)&pow_fb_dev,2*sizeof(hipfftReal)*fft_per_chunk) );
  kur_dev = pow_dev + nkurto_per_chunk;
  kur_fb_dev = pow_fb_dev + fft_per_chunk;

  // store D'Agostino statistic for thresholding
  // only using one per pol now, but keep memory size for both pols
  // to make life easier; the values are duplicated
  hipfftReal* dag_dev=NULL;
  hipMalloc ((void**)&dag_dev,sizeof(hipfftReal)*nkurto_per_chunk);
  hipfftReal* dag_fb_dev=NULL;
  hipMalloc ((void**)&dag_fb_dev,sizeof(hipfftReal)*fft_per_chunk);

  // store a set of to re-normalize voltages after applying kurtosis
  hipfftReal* kur_weights_dev=NULL;
  hipMalloc ((void**)&kur_weights_dev,sizeof(hipfftReal)*fft_per_chunk);

  // NB, reduce by a further factor of 2 if pscrunching
  int polfac = 2;
  int scrunch = (fft_per_chunk*NCHAN)/(polfac*NSCRUNCH);
  hipfftReal* fft_ave; cudacheck (
      hipMalloc ((void**)&fft_ave,sizeof(hipfftReal)*scrunch) );

  // error check that NBIT is commensurate with trimmed array size
  int trim = (fft_per_chunk*(CHANMAX-CHANMIN+1))/(polfac*NSCRUNCH);
  if (trim % (8/NBIT) != 0) {
    multilog (mlog, LOG_ERR, 
        "Selected channel and bit scheme is not commensurate!.\n");
    exit (EXIT_FAILURE);
  }

  // reduce array size by packing of samples into byte
  trim /= (8/NBIT);
  unsigned char* fft_trim_u_dev; cudacheck (
      hipMalloc ((void**)&fft_trim_u_dev,trim) );
  unsigned char* fft_trim_u_hst;

  // memory for a 10s buffer of output filterbank data
  int output_buf_sec = 10;
  int output_buf_seg_size = trim;
  int output_buf_size = output_buf_seg_size*output_buf_sec*SEG_PER_SEC;
  unsigned char* output_buf_mem;
  cudacheck (hipHostMalloc ((void**)&output_buf_mem,output_buf_size) );
  unsigned char* output_buf_cur = output_buf_mem;

  // memory for running bandpass correction; 2 pol * NCHAN
  hipfftReal* bp_dev; cudacheck (
      hipMalloc ((void**)&bp_dev,sizeof(hipfftReal)*NCHAN*2));
  cudacheck (hipMemset (bp_dev, 0, sizeof(hipfftReal)*NCHAN*2));

  // memory for FRB injection
  float* frb_delays_dev = NULL;
  cudacheck (hipMalloc ((void**)&frb_delays_dev, sizeof(float)*NCHAN));

  // prepare RNG
  hiprandGenerator_t       cugen;
  curandcheck (
      hiprandCreateGenerator (&cugen, HIPRAND_RNG_PSEUDO_DEFAULT)
      );
  curandcheck (
      hiprandSetPseudoRandomGeneratorSeed (cugen, seed)
      );


#if PROFILE
  CUDA_PROFILE_STOP(start,stop,&alloc_time)
#endif

    // constants for bandpass normalization: tsamp/tsmooth, giving a time
    // constant of about tsmooth secs.
    double tsmooth = 1;
  double tsamp = double(NFFT)/VLITE_RATE*NSCRUNCH;
  float bp_scale = tsamp/tsmooth;

  /*
  // connect to control socket
  Connection conn;
  conn.sockoptval = 1; //release port immediately after closing connection
  if (port) {
  if (serve (port, &conn) < 0) {
  multilog (mlog, LOG_ERR,
  "Failed to create control socket on port %d.\n", port);
  exit (EXIT_FAILURE);
  }
  fcntl (conn.rqst, F_SETFL, O_NONBLOCK); // set up for polling
  }
  char cmd_buff[32];
   */

  // connect to multicast injection socket
#define MAX_INJECTIONS 1
  int mc_injection_sock = open_mc_socket (mc_injectgrp, MC_INJECT_PORT,
      (char*)"Injection Socket [GENFB]", NULL, mlog);
  int injection = 0;
  injection_t ip_par;

  int quit = 0;

  double integrated  = 0.0;
  int integrated_sec = 0;

  /******************************
   **
   All injections last for 10s.
   Two seconds of noise in the beginning.
   And 8s of dispersed signals. 
   All triggers should have toa as 
   (observation_i0 + EARLY (2) ) % 10
   ******************************/

  // This point start to loop over commands.
  // 1 obs --> 2 mins --> 15 frbs of 8seconds each
  // 30 obs --> 1 hr  --> 450 frbs
  for (int iobs = 0; iobs < 50; iobs++)
  {
    // update MJD and UTCEPOCH
    UTCEPOCH += integrated_sec;
    MJD += (integrated_sec/86400.0f);
    integrated = 0.0;
    integrated_sec = 0;
    
    // add buffer space 
    // this buffer is to separate the observations
    //UTCEPOCH += 8;
    //MJD += (8.0/86400.0f);

    // do the pleasantaries
    // since only one observation happening
    // Open up filterbank file at appropriate time
    char fbfile[256];
    get_fbfile (fbfile, 256);
    uint64_t fb_bytes_written = 0;
    FILE * fb_fp = NULL;
    if (write_fb) {
      fb_fp = myopen (fbfile, "wb", true, trim);
      multilog (mlog, LOG_INFO,
          "Writing filterbank to %s.\n",fbfile);
    }

#if PROFILE
    hipEventRecord(start,0);
#endif

    if (key_out) {
      write_psrdada_header (hdu_out, fbfile);
      fprintf (stderr, "write psrdada header\n");
    }

    // write out a sigproc header
    if (write_fb)
      write_sigproc_header (fb_fp);

#if PROFILE
    CUDA_PROFILE_STOP(start,stop,&hdr_time);
#endif
    if (quit)
      break;


    // 15 frbs
    // 8s per frb
    // one observation is two minute
    // LCM (24, 60) is 120
    // 24 <- heimdall good gulp
    // 60 seconds in a minute
    for (int ifrb = 0; ifrb < 15; ifrb++) {
      // check for injection par
      // begin only if received
      int nbytes = 0;
      while (true) {
        nbytes = MultiCastReceive (mc_injection_sock, (char*)&ip_par,
            sizeof(injection_t), 0);
        multilog (mlog, LOG_INFO, "Received %d bytes.\n", nbytes);
        // to avoid a lot of IO
        if (nbytes < 0) {
          sleep (5);
          continue;
        }
        else {
          break;
        }
        // we only work with one injection
      }

      multilog (mlog, LOG_INFO, 
          "Received injection request with DM=%3.2f, Width[units]=%d" 
          " Amplitude=%5.2ef\n",
          ip_par.dm, ip_par.wd, ip_par.amp);

      // beginning new observation
      hipEventRecord(obs_start,0);

      set_frb_delays <<< NCHAN/NTHREAD+1, NTHREAD >>> (frb_delays_dev, ip_par.dm);
      cudacheck (hipGetLastError () );

      /*
         float* frb_delays_hst = NULL;
         cudacheck (hipHostMalloc ((void**)&frb_delays_hst, sizeof(float)*NCHAN));
         cudacheck (hipMemcpy (

         frb_delays_hst,frb_delays_dev,sizeof(float)*NCHAN,
         hipMemcpyDeviceToHost) );
         for (int ichan=0; ichan < 6250; ichan += 100)
         fprintf (stdout, "frb_delay %d = %.6f\n", ichan, frb_delays_hst[ichan]);
       */

#if RT_PROFILE
      hipEventRecord(rt_start,0);
      measured_time = 0;
#endif 

      int SECS = 8;

      for (int isec = 0; isec < SECS; isec++) // loop over segments
      {
        // We have a 10s buffer on host
        // which we cycle through.
        if ( (integrated_sec % output_buf_sec) == 0 )
          output_buf_cur = output_buf_mem;
        // do dispatch -- break into chunks to fit in GPU memory; this is
        // currently 100 milliseconds
        for (int iseg = 0; iseg < SEG_PER_SEC; iseg++)
        {

          ////// FILL WITH WHITE NOISE   //////
#if PROFILE
          hipEventRecord (start,0);
#endif 

          cudacheck (hipGetLastError () );

          // fill fft_in with samps_per_chunk white noise
          curandcheck (
              hiprandGenerateNormal (
                cugen, fft_in, samps_per_chunk,
                0.0f, 33.818f
                )
              );

#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            convert_time += elapsed;
#endif

          ////// CALCULATE KURTOSIS STATISTICS //////
#if PROFILE
          hipEventRecord (start,0);
#endif

          // calculate high time resolution kurtosis (250 or 500 samples)
          kurtosis <<<nkurto_per_chunk, 256>>> (
              fft_in,pow_dev,kur_dev);
          cudacheck (hipGetLastError () );

          // compute the thresholding statistic
          // NB now modified to combine polarizations
          compute_dagostino <<<nsms*32,NTHREAD>>> (
              kur_dev,dag_dev,nkurto_per_chunk/2);
          cudacheck (hipGetLastError () );

          // calculate coarser kurtosis (for entire filterbank sample, e.g. 12500 samples)
          // NB this relies on results of previous D'Agostino calculation
          block_kurtosis <<<fft_per_chunk/8,256>>> (
              pow_dev,kur_dev,dag_dev,pow_fb_dev,kur_fb_dev);
          cudacheck (hipGetLastError () );
          // NB now modified to combine polarizations
          compute_dagostino2 <<<nsms*32,NTHREAD>>> (
              kur_fb_dev,dag_fb_dev,fft_per_chunk/2);
          cudacheck (hipGetLastError () );

          cudacheck (hipMemset (
                kur_weights_dev,0,sizeof(hipfftReal)*fft_per_chunk) );
          // (1) NB that fft_in_kur==fft_in if not writing both streams
          // (2) original implementation had a block for each pol; keeping
          //     that, but two blocks now access the same Dagostino entry
          //if (integrated >= 0.1)
          apply_kurtosis <<<nkurto_per_chunk, 256>>> (
              fft_in,fft_in_kur,dag_dev,dag_fb_dev,kur_weights_dev);
          cudacheck (hipGetLastError () );

#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            kurtosis_time += elapsed;
#endif

          ////// PERFORM FFTs //////
#if PROFILE
          hipEventRecord (start,0);
#endif 
          cufftcheck (hipfftExecR2C (plan,fft_in_kur,fft_out) );
#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            fft_time += elapsed;
#endif

          ////// INJECT FRB AS REQUESTED //////
          if (1)
          {
            // NB that inject_frb_now is only reset every 1s, so we also use
            // it to keep track of how many segments have elapsed since the
            // FRB time, since this loop is over 100ms chunks which will be
            // < dispersed FRB width, typically

            int since_frb = ( ( isec *SEG_PER_SEC ) + iseg-1 )*FFTS_PER_SEG;
            inject_frb <<< NCHAN/NTHREAD+1,NTHREAD >>> (fft_out, 
                frb_delays_dev, since_frb,
                ip_par.wd, ip_par.amp
                );
            cudacheck ( hipGetLastError () );
          }

          ////// NORMALIZE BANDPASS //////
#if PROFILE
          hipEventRecord(start,0);
#endif 
          detect_and_normalize3 <<<(NCHAN*2)/NTHREAD+1,NTHREAD>>> (
              fft_out,kur_weights_dev,bp_dev,bp_scale);
          cudacheck ( hipGetLastError () );
#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            normalize_time += elapsed;
#endif

          ////// ADD POLARIZATIONS //////
#if PROFILE
          hipEventRecord (start,0);
#endif 
          maxn = (fft_per_chunk*NCHAN)/polfac;
          pscrunch_weights <<<nsms*32,NTHREAD>>> (
              fft_out,kur_weights_dev,maxn);
          cudacheck ( hipGetLastError () );
#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            pscrunch_time += elapsed;
#endif

          ////// AVERAGE TIME DOMAIN //////
#if PROFILE
          hipEventRecord( start,0);
#endif 
          maxn /= NSCRUNCH;
          tscrunch <<<nsms*32,NTHREAD>>> (fft_out,fft_ave,maxn);
          cudacheck ( hipGetLastError () );
#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            tscrunch_time += elapsed;
#endif

          ////// TRIM CHANNELS AND DIGITIZE //////
#if PROFILE
          hipEventRecord (start,0);
#endif 
          maxn = (CHANMAX-CHANMIN+1)*(maxn/NCHAN)/(8/NBIT);
          switch (NBIT)
          {
            case 2:
              sel_and_dig_2b <<<nsms*32,NTHREAD>>> (
                  fft_ave,fft_trim_u_dev,maxn, npol);
              break;
            case 4:
              sel_and_dig_4b <<<nsms*32,NTHREAD>>> (
                  fft_ave,fft_trim_u_dev,maxn, npol);
              break;
            case 8:
              sel_and_dig_8b <<<nsms*32,NTHREAD>>> (
                  fft_ave,fft_trim_u_dev,maxn, npol);
              break;
            default:
              sel_and_dig_8b <<<nsms*32,NTHREAD>>> (
                  fft_ave,fft_trim_u_dev,maxn, npol);
              break;
          }
          cudacheck ( hipGetLastError () );
#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            digitize_time += elapsed;
#endif

#if PROFILE
          hipEventRecord (start,0);
#endif 

          // copy filterbanked data back to host; use big buffer to avoid
          // a second copy; NB that if we are only recording a single RFI
          // excision mode, the _kur buffer points to same place.  And if we
          // are recording both, then we output the kurtosis.  So we can
          // just always record the kurtosis to the output buffer, and if
          // we are recording both, copy the second to the small buffer
          fft_trim_u_hst = output_buf_cur;
          cudacheck (hipMemcpy (
                fft_trim_u_hst,fft_trim_u_dev,maxn,hipMemcpyDeviceToHost) );
          output_buf_cur += output_buf_seg_size; 
          cudacheck (hipGetLastError () );

#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            misc_time += elapsed;
#endif

          // finally, push the filterbanked time samples onto psrdada buffer
          // and/or write out to sigproc

#if PROFILE
          hipEventRecord (start,0);
#endif 

          char* outbuff = (char*)fft_trim_u_hst;
          // check_buffer (hdu_out, mlog);
          if (key_out) {
            check_ipcio_write (hdu_out, outbuff, maxn, mlog);
          }

          // TODO -- tune this I/O.  The buffer size is set to 8192, but
          // according to fstat the nfs wants a block size of 1048576! Each
          // 100ms of data is 65536 with the current parameters.  So optimally
          // we would buffer in memory for the full 1 second before a write.
          // However, a simple improvement will be reducing the write calls by
          // a factor of 8 by either changing the buffer size or using the

          // TODO -- add error checking for these writes
          if (write_fb) {
            fwrite (fft_trim_u_hst,1,maxn,fb_fp);
            fb_bytes_written += maxn;
          }

#if PROFILE
          CUDA_PROFILE_STOP(start,stop,&elapsed)
            write_time += elapsed;
#endif

          integrated += 1./double(SEG_PER_SEC);

        } // end loop over segments

        cudacheck (hipGetLastError () );

        // sleep for 1.0
        sleep (1);

        integrated_sec += 1;
      } // end loop over seconds
    } // end loop over frb

    cudacheck ( hipGetLastError () );
    if (key_out) {
      fprintf (stderr, "genfb: before dada_hdu_unlock_write\n");
      dadacheck (dada_hdu_unlock_write (hdu_out));
      fprintf (stderr, "genfb: after dada_hdu_unlock_write\n");
    }
    fflush (stderr);

#if PROFILE
    hipEventRecord(start,0);
#endif 

    // close files
    if (write_fb) {
      if (fb_fp) {fclose (fb_fp); fb_fp = NULL;}
      uint64_t samps_written = (fb_bytes_written*(8/NBIT))/(CHANMAX-CHANMIN+1);
      multilog (mlog, LOG_INFO, "Wrote %.2f MB (%.2f s) to %s\n",
          fb_bytes_written*1e-6,samps_written*tsamp,fbfile);
    }

    float obs_time;
    CUDA_PROFILE_STOP (obs_start,obs_stop,&obs_time);
    multilog (mlog, LOG_INFO, "Proc Time...%.3f\n", obs_time*1e-3);

#if PROFILE
    CUDA_PROFILE_STOP(start,stop,&flush_time)
      float sub_time = hdr_time + read_time + todev_time + 
      histo_time + convert_time + kurtosis_time + fft_time + 
      normalize_time + pscrunch_time + tscrunch_time + digitize_time + 
      write_time + flush_time + misc_time;
    multilog (mlog, LOG_INFO, "Alloc Time..%.3f\n", alloc_time*1e-3);
    multilog (mlog, LOG_INFO, "Histogram...%.3f\n", histo_time*1e-3);
    multilog (mlog, LOG_INFO, "Convert.....%.3f\n", convert_time*1e-3);
    multilog (mlog, LOG_INFO, "Kurtosis....%.3f\n", kurtosis_time*1e-3);
    multilog (mlog, LOG_INFO, "FFT.........%.3f\n", fft_time*1e-3);
    multilog (mlog, LOG_INFO, "Normalize...%.3f\n", normalize_time*1e-3);
    multilog (mlog, LOG_INFO, "Pscrunch....%.3f\n", pscrunch_time*1e-3);
    multilog (mlog, LOG_INFO, "Tscrunch....%.3f\n", tscrunch_time*1e-3);
    multilog (mlog, LOG_INFO, "Digitize....%.3f\n", digitize_time*1e-3);
    multilog (mlog, LOG_INFO, "Write.......%.3f\n", write_time*1e-3);
    multilog (mlog, LOG_INFO, "Flush.......%.3f\n", flush_time*1e-3);
    multilog (mlog, LOG_INFO, "Misc........%.3f\n", misc_time*1e-3);
    multilog (mlog, LOG_INFO, "Sum of subs.%.3f\n", sub_time*1e-3);

    // reset values for next loop
    hdr_time=read_time=todev_time=convert_time=kurtosis_time=fft_time=0;
    histo_time=normalize_time=tscrunch_time=pscrunch_time=digitize_time=0;
    write_time=flush_time=misc_time=elapsed=0;

#endif

    // sleep for 3 seconds after every observation
    sleep (3);

  } // end loop over observations

  // ffts
  if (fft_in) hipFree (fft_in);
  if (fft_in_kur) hipFree (fft_in_kur);
  if (fft_out) hipFree (fft_out);
  // kur
  if (pow_dev) hipFree (pow_dev);
  if (kur_dev) hipFree (kur_dev);
  if (dag_dev) hipFree (dag_dev);
  if (dag_fb_dev) hipFree (dag_fb_dev);
  if (kur_weights_dev) hipFree (kur_weights_dev);
  // pscrunch
  if (fft_ave) hipFree (fft_ave);
  // digitized
  if (fft_trim_u_dev) hipFree (fft_trim_u_dev);
  if (fft_trim_u_hst) hipHostFree (fft_trim_u_hst);
  if (output_buf_mem ) hipHostFree (output_buf_mem );
  // bandpass
  if (bp_dev) hipFree (bp_dev);
  // frb delays
  if (frb_delays_dev) hipFree (frb_delays_dev);
  // RNG
  hiprandDestroyGenerator (cugen);

  return exit_status;

}

